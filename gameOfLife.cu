#include "hip/hip_runtime.h"
/*
Game of Life ticker from text.
Copyright (C) 2016  Brad Parker Keserich

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include "gameOfLife.h"

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define THREADS_PER_BLOCK 512

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__constant__ WorldData g_worldData;

//TODO should do something similar to stencils
__global__ void gol_deviceUpdate(GLuint * prevColor, GLuint * nextColor) 
{
    uint index = threadIdx.x + blockIdx.x * blockDim.x;
    uint width = g_worldData.width;
    uint height = g_worldData.height;
    uint numCells = width * height;
    
    if (index >= numCells) return; // disallow indices too high
    
    uint x = index / height;
    uint y = index % height;
    uint x_m1 = (x - 1) % width;
    if (x - 1 == -1) x_m1 = (width - 1); // for correct modulus
    uint x_p1 = (x + 1) % width;
    uint y_m1 = (y - 1) % height;
    if (y - 1 == -1) y_m1 = (height - 1); // for correct modulus
    uint y_p1 = (y + 1) % height;
    bool s = (1 == prevColor[2 * index]); // current state //TODO make sure that we have only 1 and 0 values for prevColor
    uint n = prevColor[(2 * (height * x_m1 + y_m1)) % (2 * numCells)] //TODO redundant calculations here
             + prevColor[(2 * (height * x + y_m1)) % (2 * numCells)]
             + prevColor[(2 * (height * x_p1 + y_m1)) % (2 * numCells)]
             + prevColor[(2 * (height * x_m1 + y)) % (2 * numCells)]
             //+ prevColor[2 * (index + x + y)]
             + prevColor[(2 * (height * x_p1 + y)) % (2 * numCells)]
             + prevColor[(2 * (height * x_m1 + y_p1)) % (2 * numCells)]
             + prevColor[(2 * (height * x + y_p1)) % (2 * numCells)]
             + prevColor[(2 * (height * x_p1 + y_p1)) % (2 * numCells)];
    
    if (s && n < 2)
    {
        nextColor[2 * index] = 0;// dead
        nextColor[2 * index + 1] = 0;
    }
    else if (s && (n == 2 || n == 3))
    {
        nextColor[2 * index] = 1;// alive
        nextColor[2 * index + 1] = 1;
    }
    else if (s && n > 3)
    {
        nextColor[2 * index] = 0;// dead
        nextColor[2 * index + 1] = 0;
    }
    else if (!s && n == 3)
    {
        nextColor[2 * index] = 1;// alive
        nextColor[2 * index + 1] = 1;
    }
    else
    {
        nextColor[2 * index] = prevColor[2 * index];// previous
        nextColor[2 * index + 1] = prevColor[2 * index + 1];
    }
    nextColor[2 * index + 1] = 3;// = n;
}

__global__ void gol_deviceInit(GLuint * prevColor, GLuint * nextColor) //TODO would this be better as a flat out memcpy instead?
{
    //TODO could reduce the number of calls later by tracking live cell locations
    //TODO a memcpy of some sort on device might be faster here
    uint index = threadIdx.x + blockIdx.x * blockDim.x;
    prevColor[2 * index] = nextColor[2 * index];
    prevColor[2 * index + 1] = nextColor[2 * index + 1];
}

__global__ void gol_test()
{ 
    uint index = threadIdx.x + blockIdx.x * blockDim.x;
    printf("test index %i\n", index); //TODO verbose
}

GameOfLife::GameOfLife(uint width, uint height, bool data[])
{
    _initialize(width, height, data, false);
}

GameOfLife::GameOfLife(uint width, uint height, bool data[], bool useOpenGL)
{
    _initialize(width, height, data, useOpenGL);
}

GameOfLife::~GameOfLife()
{
    delete[] m_colorData;
    hipFree(d_colorData);
    if (m_useOpenGL)
        cudaGLUnregisterBufferObject(m_colorBuffer);
    else
        hipFree(d_nextColorData);
}

void GameOfLife::_initialize(uint width, uint height, bool data[], bool useOpenGL)
{
    printf("width = %i, height = %i, total = %i\n", width, height, width * height); //TODO verbose
    m_worldData.width = width;
    m_worldData.height = height;
    m_useOpenGL = useOpenGL;
    
    m_numThreads = 4 * 32; //TODO could be setting better
    m_numBlocks = ((width * height) / m_numThreads) + 1; // N / THREADS_PER_BLOCK
    printf("m_numBlocks = %i, m_numThreads = %i, total = %i\n", m_numBlocks, m_numThreads, m_numBlocks * m_numThreads); //TODO verbose
    m_colorData = new GLuint[2 * width * height];
    m_tmpColorData = new GLuint[2 * width * height]; //TODO for debugging
    for (uint j = 0; j < height; j++)
    {
        for (uint i = 0; i < width; i++)
        {
            if (data[height * i + j])
            {
                m_colorData[2 * height * i + 2 * j] = 1;
                m_colorData[2 * height * i + 2 * j + 1] = 1;
            }
            else
            {
                m_colorData[2 * height * i + 2 * j] = 0;
                m_colorData[2 * height * i + 2 * j + 1] = 0;
            }
            m_tmpColorData[2 * height * i + 2 * j] = m_colorData[2 * height * i + 2 * j]; //TODO for debug
            m_tmpColorData[2 * height * i + 2 * j + 1] = m_colorData[2 * height * i + 2 * j + 1]; //TODO for debug
            //}
        }
    }
    
    hipMalloc((void **) &d_colorData, 2 * width * height * sizeof(GLuint)); // allocate for prevColor to be held
    if (!m_useOpenGL) hipMemcpy(d_nextColorData, m_colorData, 2 * width * height * sizeof(GLuint), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(g_worldData), &m_worldData, sizeof(WorldData));
}

void GameOfLife::update() 
{
    if (m_useOpenGL)
    {
        void* ptr;
        cudaGLMapBufferObject(&ptr, m_colorBuffer);
        gol_deviceInit<<<m_numBlocks, m_numThreads>>>(d_colorData, (GLuint *)ptr); //TODO need to define prevColorBuffer
        gol_deviceUpdate<<<m_numBlocks, m_numThreads>>>(d_colorData, (GLuint *)ptr);
    
        //gpuErrchk( hipPeekAtLastError() );
        //gpuErrchk( hipDeviceSynchronize() );
    
        //hipMemcpy(m_colorData, d_colorData, 2 * m_worldData.width * m_worldData.height * sizeof(GLuint), hipMemcpyDeviceToHost); //TODO debugging
        cudaGLUnmapBufferObject(m_colorBuffer);
    }
    else
    {
        gol_deviceInit<<<m_numBlocks, m_numThreads>>>(d_colorData, d_nextColorData); //TODO need to define prevColorBuffer
        gol_deviceUpdate<<<m_numBlocks, m_numThreads>>>(d_colorData, d_nextColorData);
    }
} 

GLuint * GameOfLife::getColorBuffer()
{
    return &m_colorBuffer;
}

GLuint * GameOfLife::getColorData()
{
    return m_colorData;
}

WorldData * GameOfLife::getWorldData()
{
    return &m_worldData;
}

void GameOfLife::retrieveRLEData(RLE * rle)
{
    //TODO determine if we need to use the cudaGLMapBufferObject and cudaGLUnmapBufferObject calls
    if (m_useOpenGL)
    {
        void* ptr;
        cudaGLMapBufferObject(&ptr, m_colorBuffer);
        //gol_deviceInit<<<m_numBlocks, m_numThreads>>>(d_colorData, (GLuint *)ptr);  //TODO necessary?
        hipMemcpy(m_colorData, d_colorData, 2 * m_worldData.width * m_worldData.height * sizeof(GLuint), hipMemcpyDeviceToHost);
        cudaGLUnmapBufferObject(m_colorBuffer);
    }
    else
    {
        hipMemcpy(m_colorData, d_colorData, 2 * m_worldData.width * m_worldData.height * sizeof(GLuint), hipMemcpyDeviceToHost);
    }
    
    //TODO if rle is nullptr or points to garbage, then error out
    //TODO should we check the old data and delete it if it is not yet set to nullptr?
    
    rle->x = m_worldData.width;
    rle->y = m_worldData.height;
    rle->data = new bool[rle->x * rle->y];
    
    for (uint j = 0; j < m_worldData.height; j++)
    {
        for (uint i = 0; i < m_worldData.width; i++)
        {
            if (m_colorData[2 * m_worldData.height * i + 2 * j] == 1)
                rle->data[m_worldData.height * i + j] = true;
            else
                rle->data[m_worldData.height * i + j] = false;
        }
    }
    
    printf("Neighbor Data from GPU:\n"); //TODO verbose
    for (uint j = 0; j < m_worldData.height; j++)
    {
        for (uint i = 0; i < m_worldData.width; i++)
        {
            printf("%s ",std::to_string(m_colorData[2 * m_worldData.height * i + 2 * j + 1]).c_str());
        }
        printf("\n");
    }
}
